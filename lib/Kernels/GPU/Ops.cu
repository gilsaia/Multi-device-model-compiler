#include "multi-device-model-compiler/Kernels/GPU/Ops.h"

#include "llvm/ADT/DenseMap.h"
#include "llvm/ADT/Hashing.h"
#include "llvm/Support/raw_ostream.h"

#include "hipblaslt.h"

#include <stdexcept>

static hipblasLtHandle_t ltHandle;

static void *workspace;
static size_t workspaceSize = 1024 * 1024 * 16;

inline void checkCudaStatus(hipError_t status) {
  if (status != hipSuccess) {
    printf("cuda API failed with status %d: %s\n", status,
           hipGetErrorString(status));
    throw std::logic_error("cuda API failed");
  }
}

inline void checkCublasStatus(hipblasStatus_t status) {
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("cuBLAS API failed with status %d\n", status);
    throw std::logic_error("cuBLAS API failed");
  }
}

namespace llvm {
struct SmallVectorHasher {
  std::size_t operator()(const SmallVector<int64_t> &vec) const {
    return hash_combine_range(vec.begin(), vec.end());
  }
};
template <> struct DenseMapInfo<SmallVector<int64_t>> {
  static inline SmallVector<int64_t> getEmptyKey() {
    SmallVector<int64_t> EmptyKey{-1};
    return EmptyKey;
  }
  static inline SmallVector<int64_t> getTombstoneKey() {
    SmallVector<int64_t> TombstoneKey{-2};
    return TombstoneKey;
  }
  static unsigned getHashValue(const SmallVector<int64_t> &vec) {
    return SmallVectorHasher()(vec);
  }
  static bool isEqual(const SmallVector<int64_t> &lhs,
                      const SmallVector<int64_t> &rhs) {
    return lhs == rhs;
  }
};
} // namespace llvm

void gpuOpsInit() {
  checkCublasStatus(hipblasLtCreate(&ltHandle));
  checkCudaStatus(hipMalloc(&workspace, workspaceSize));
}

void gpuOpsDeinit() {
  checkCublasStatus(hipblasLtDestroy(ltHandle));
  checkCudaStatus(hipFree(workspace));
}

static llvm::DenseMap<llvm::SmallVector<int64_t>,
                      hipblasLtMatmulHeuristicResult_t>
    matmulMap;

extern "C" MLIR_GPU_OPS_EXPORT void mgpuMatmul(float *input, float *weight,
                                               float *bias, float *output,
                                               int64_t M, int64_t N, int64_t K,
                                               hipStream_t stream) {
  llvm::SmallVector<int64_t> matmulKey{M, N, K};
  hipblasLtMatmulDesc_t operationDesc = NULL;
  hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Ddesc = NULL;

  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtMatmulPreference_t preference = NULL;
  int returnedResults = 0;

  auto epilogue = HIPBLASLT_EPILOGUE_BIAS;
  checkCublasStatus(hipblasLtMatmulDescCreate(
      &operationDesc, HIPBLAS_COMPUTE_32F_FAST_TF32, HIP_R_32F));
  checkCublasStatus(hipblasLtMatmulDescSetAttribute(
      operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue,
      sizeof(epilogue)));
  checkCublasStatus(hipblasLtMatmulDescSetAttribute(
      operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias,
      sizeof(float *)));

  checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, N, K, N));
  checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, K, M, K));
  checkCublasStatus(hipblasLtMatrixLayoutCreate(&Ddesc, HIP_R_32F, N, M, N));
  if (matmulMap.count(matmulKey)) {
    heuristicResult = matmulMap[matmulKey];
  } else {
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(
        preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize,
        sizeof(workspaceSize)));

    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(
        ltHandle, operationDesc, Adesc, Bdesc, Ddesc, Ddesc, preference, 1,
        &heuristicResult, &returnedResults));

    if (returnedResults == 0) {
      checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);
    }
    matmulMap.insert({matmulKey, heuristicResult});
  }

  float alpha = 1, beta = 0;
  checkCublasStatus(hipblasLtMatmul(ltHandle, operationDesc, &alpha, weight,
                                   Adesc, input, Bdesc, &beta, output, Ddesc,
                                   output, Ddesc, &heuristicResult.algo,
                                   workspace, workspaceSize, stream));

  if (preference)
    checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
  if (Ddesc)
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(Ddesc));
  if (Bdesc)
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
  if (Adesc)
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
  if (operationDesc)
    checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));
}