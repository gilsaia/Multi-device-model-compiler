#include "hip/hip_runtime.h"
#include "multi-device-model-compiler/Kernels/GPU/Ops.h"

#include "llvm/ADT/DenseMap.h"
#include "llvm/ADT/Hashing.h"
#include "llvm/Support/raw_ostream.h"

#include "hipblaslt.h"
#include "hipDNN.h"

#include "xxhash.h"

#include <float.h>
#include <stdexcept>

static hipblasLtHandle_t ltHandle;
static hipdnnHandle_t cudnnHandle;

static void *workspace;
static size_t workspaceSize = 1024 * 1024 * 16;

inline void checkCudaStatus(hipError_t status) {
  if (status != hipSuccess) {
    printf("cuda API failed with status %d: %s\n", status,
           hipGetErrorString(status));
    throw std::logic_error("cuda API failed");
  }
}

inline void checkCublasStatus(hipblasStatus_t status) {
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("cuBLAS API failed with status %d\n", status);
    throw std::logic_error("cuBLAS API failed");
  }
}

inline void checkCudnnStatus(hipdnnStatus_t status) {
  if (status != HIPDNN_STATUS_SUCCESS) {
    printf("cuDNN API failed with status %d\n", status);
    throw std::logic_error("cuDNN API failed");
  }
}

void gpuOpsInit() {
  checkCublasStatus(hipblasLtCreate(&ltHandle));
  checkCudaStatus(hipMalloc(&workspace, workspaceSize));
  checkCudnnStatus(hipdnnCreate(&cudnnHandle));
}

void gpuOpsDeinit() {
  checkCudnnStatus(hipdnnDestroy(cudnnHandle));
  checkCublasStatus(hipblasLtDestroy(ltHandle));
  checkCudaStatus(hipFree(workspace));
}

static std::unordered_map<int64_t, hipblasLtMatmulHeuristicResult_t> matmulMap;

extern "C" MLIR_GPU_OPS_EXPORT void mgpuMatmul(float *input, float *weight,
                                               float *bias, float *output,
                                               int64_t M, int64_t N, int64_t K,
                                               hipStream_t stream) {
  // llvm::SmallVector<int64_t> matmulKey{M, N, K};
  std::array<int64_t, 3> keys{M, N, K};
  int64_t matmulKey = XXH3_64bits(keys.data(), keys.size() * sizeof(int64_t));
  hipblasLtMatmulDesc_t operationDesc = NULL;
  hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Ddesc = NULL;

  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtMatmulPreference_t preference = NULL;
  int returnedResults = 0;

  auto epilogue = HIPBLASLT_EPILOGUE_BIAS;
  checkCublasStatus(hipblasLtMatmulDescCreate(
      &operationDesc, HIPBLAS_COMPUTE_32F_FAST_TF32, HIP_R_32F));
  checkCublasStatus(hipblasLtMatmulDescSetAttribute(
      operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue,
      sizeof(epilogue)));
  checkCublasStatus(hipblasLtMatmulDescSetAttribute(
      operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias,
      sizeof(float *)));

  checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, N, K, N));
  checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, K, M, K));
  checkCublasStatus(hipblasLtMatrixLayoutCreate(&Ddesc, HIP_R_32F, N, M, N));
  if (matmulMap.count(matmulKey)) {
    heuristicResult = matmulMap[matmulKey];
  } else {
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(
        preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize,
        sizeof(workspaceSize)));

    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(
        ltHandle, operationDesc, Adesc, Bdesc, Ddesc, Ddesc, preference, 1,
        &heuristicResult, &returnedResults));

    if (returnedResults == 0) {
      checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);
    }
    matmulMap.emplace(matmulKey, heuristicResult);
  }

  float alpha = 1, beta = 0;
  checkCublasStatus(hipblasLtMatmul(ltHandle, operationDesc, &alpha, weight,
                                   Adesc, input, Bdesc, &beta, output, Ddesc,
                                   output, Ddesc, &heuristicResult.algo,
                                   workspace, workspaceSize, stream));

  if (preference)
    checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
  if (Ddesc)
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(Ddesc));
  if (Bdesc)
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
  if (Adesc)
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
  if (operationDesc)
    checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));
}

static std::unordered_map<int64_t, hipdnnConvolutionFwdAlgo_t> conv2dMap;

extern "C" MLIR_GPU_OPS_EXPORT void
mgpuConv2d(float *input, float *weight, float *bias, float *output,
           float *postAdd, int64_t N, int64_t IC, int64_t H, int64_t W,
           int64_t OC, int64_t KH, int64_t KW, int64_t OH, int64_t OW,
           int64_t PHL, int64_t PWL, int64_t PHR, int64_t PWR, int64_t SH,
           int64_t SW, int64_t DH, int64_t DW, bool hasPostAdd,
           bool hasContainRelu, hipStream_t stream) {
  std::array<int64_t, 10> keys{N,   IC, H,  OC,         KH,
                               PHL, SH, DH, hasPostAdd, hasContainRelu};
  int64_t convKey = XXH3_64bits(keys.data(), keys.size() * sizeof(int64_t));

  hipdnnSetStream(cudnnHandle, stream);

  hipdnnConvolutionDescriptor_t convDesc;
  checkCudnnStatus(hipdnnCreateConvolutionDescriptor(&convDesc));
  checkCudnnStatus(hipdnnSetConvolution2dDescriptor(
      convDesc, PHL, PWL, SH, SW, DH, DW, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));
  checkCudnnStatus(hipdnnSetConvolutionMathType(
      convDesc, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));

  hipdnnTensorDescriptor_t inputDesc, biasDesc, outputDesc, addDesc;
  float *add;
  checkCudnnStatus(hipdnnCreateTensorDescriptor(&inputDesc));
  checkCudnnStatus(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW,
                                              HIPDNN_DATA_FLOAT, N, IC, H, W));
  checkCudnnStatus(hipdnnCreateTensorDescriptor(&biasDesc));
  checkCudnnStatus(hipdnnSetTensor4dDescriptor(biasDesc, HIPDNN_TENSOR_NCHW,
                                              HIPDNN_DATA_FLOAT, 1, OC, 1, 1));
  checkCudnnStatus(hipdnnCreateTensorDescriptor(&outputDesc));
  checkCudnnStatus(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW,
                                              HIPDNN_DATA_FLOAT, N, OC, OH, OW));
  if (hasPostAdd) {
    checkCudnnStatus(hipdnnCreateTensorDescriptor(&addDesc));
    checkCudnnStatus(hipdnnSetTensor4dDescriptor(
        addDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, OC, OH, OW));
    add = postAdd;
  } else {
    addDesc = outputDesc;
    add = output;
  }

  hipdnnFilterDescriptor_t weightDesc;
  checkCudnnStatus(hipdnnCreateFilterDescriptor(&weightDesc));
  checkCudnnStatus(hipdnnSetFilter4dDescriptor(
      weightDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, OC, IC, KH, KW));

  hipdnnActivationDescriptor_t actDesc;
  checkCudnnStatus(hipdnnCreateActivationDescriptor(&actDesc));
  checkCudnnStatus(hipdnnSetActivationDescriptor(
      actDesc,
      hasContainRelu ? HIPDNN_ACTIVATION_RELU : HIPDNN_ACTIVATION_PATHTRU,
      HIPDNN_NOT_PROPAGATE_NAN, DBL_MAX));

  hipdnnConvolutionFwdAlgo_t algo;
  if (conv2dMap.count(convKey)) {
    algo = conv2dMap[convKey];
  } else {
    algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
    if (hasContainRelu) {
      int algoMax;
      checkCudnnStatus(
          cudnnGetConvolutionForwardAlgorithmMaxCount(cudnnHandle, &algoMax));
      std::vector<hipdnnConvolutionFwdAlgoPerf_t> perfs(algoMax);
      int algoFind;
      checkCudnnStatus(hipdnnFindConvolutionForwardAlgorithm(
          cudnnHandle, inputDesc, weightDesc, convDesc, outputDesc, algoMax,
          &algoFind, perfs.data()));
      perfs.resize(algoFind);
      if (algoFind < 1) {
        checkCudnnStatus(HIPDNN_STATUS_NOT_SUPPORTED);
      }
      algo = perfs[0].algo;
    }

    size_t nWorkSpaceSize;
    checkCudnnStatus(hipdnnGetConvolutionForwardWorkspaceSize(
        cudnnHandle, inputDesc, weightDesc, convDesc, outputDesc, algo,
        &nWorkSpaceSize));
    if (nWorkSpaceSize > workspaceSize) {
      hipFree(workspace);
      hipMalloc(&workspace, nWorkSpaceSize);
      workspaceSize = nWorkSpaceSize;
    }
    conv2dMap.emplace(convKey, algo);
  }

  float alpha1 = 1, alpha2 = hasPostAdd ? 1 : 0;

  checkCudnnStatus(cudnnConvolutionBiasActivationForward(
      cudnnHandle, &alpha1, inputDesc, input, weightDesc, weight, convDesc,
      algo, workspace, workspaceSize, &alpha2, addDesc, add, biasDesc, bias,
      actDesc, outputDesc, output));

  hipdnnDestroyConvolutionDescriptor(convDesc);
  hipdnnDestroyTensorDescriptor(inputDesc);
  hipdnnDestroyTensorDescriptor(biasDesc);
  hipdnnDestroyTensorDescriptor(outputDesc);
  hipdnnDestroyFilterDescriptor(weightDesc);
  hipdnnDestroyActivationDescriptor(actDesc);
  if (hasPostAdd) {
    hipdnnDestroyTensorDescriptor(addDesc);
  }
}

extern "C" MLIR_GPU_OPS_EXPORT void
mgpuPool2d(float *input, float *output, int64_t N, int64_t C, int64_t H,
           int64_t W, int64_t OH, int64_t OW, int64_t KH, int64_t KW,
           int64_t PHL, int64_t PWL, int64_t PHR, int64_t PWR, int64_t SH,
           int64_t SW, int64_t method /* 0 - max, 1 - avg */,
           hipStream_t stream) {
  hipdnnSetStream(cudnnHandle, stream);

  hipdnnPoolingDescriptor_t poolDesc;
  hipdnnPoolingMode_t mode = (method == 0)
                                ? HIPDNN_POOLING_MAX
                                : HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING;
  checkCudnnStatus(hipdnnCreatePoolingDescriptor(&poolDesc));
  checkCudnnStatus(hipdnnSetPooling2dDescriptor(
      poolDesc, mode, HIPDNN_NOT_PROPAGATE_NAN, KH, KW, PHL, PWL, SH, SW));

  hipdnnTensorDescriptor_t inputDesc, outputDesc;
  checkCudnnStatus(hipdnnCreateTensorDescriptor(&inputDesc));
  checkCudnnStatus(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW,
                                              HIPDNN_DATA_FLOAT, N, C, H, W));
  checkCudnnStatus(hipdnnCreateTensorDescriptor(&outputDesc));
  checkCudnnStatus(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW,
                                              HIPDNN_DATA_FLOAT, N, C, OH, OW));

  float alpha = 1, beta = 0;

  checkCudnnStatus(hipdnnPoolingForward(cudnnHandle, poolDesc, &alpha, inputDesc,
                                       input, &beta, outputDesc, output));

  hipdnnDestroyTensorDescriptor(inputDesc);
  hipdnnDestroyTensorDescriptor(outputDesc);
  hipdnnDestroyPoolingDescriptor(poolDesc);
}