#include "hip/hip_runtime.h"
#include "multi-device-model-compiler/Kernels/GPU/Utils.h"

#include "hip/hip_fp16.h"

template <size_t bytes>
__device__ void cudaReadBytes(void *dst, void const *src);

template <> __device__ void cudaReadBytes<8>(void *dst, void const *src) {
  reinterpret_cast<float2 *>(dst)[0] = reinterpret_cast<float2 const *>(src)[0];
}

template <> __device__ void cudaReadBytes<16>(void *dst, void const *src) {
  reinterpret_cast<float4 *>(dst)[0] = reinterpret_cast<float4 const *>(src)[0];
}

template <typename T_OUT, typename T_IN, size_t TPB, size_t VPT>
__global__ void cudaCast(T_OUT *dst, T_IN const *const src, const size_t size) {
  __shared__ T_IN srcTmps[TPB * VPT];
  __shared__ T_OUT dstTmps[TPB * VPT];
  for (size_t tid = (threadIdx.x + blockIdx.x * blockDim.x) * VPT; tid < size;
       tid += blockDim.x * gridDim.x * VPT) {
    cudaReadBytes<VPT * sizeof(T_IN)>(
        reinterpret_cast<void *>(srcTmps + threadIdx.x * VPT),
        reinterpret_cast<void const *>(src + tid));
    __syncwarp();
    int sharedid = threadIdx.x / 32 * (32 * VPT) + threadIdx.x % 32;
#pragma unroll
    for (int i = 0; i < VPT; ++i) {
      dstTmps[sharedid + i * 32] = (T_OUT)(srcTmps[sharedid + i * 32]);
    }
    __syncwarp();
    cudaReadBytes<VPT * sizeof(T_OUT)>(
        reinterpret_cast<void *>(dst + tid),
        reinterpret_cast<void const *>(dstTmps + threadIdx.x * VPT));
  }
}

template <size_t TPB, size_t VPT>
__global__ void cudaCastFloat2Half(half *dst, float const *const src,
                                   const size_t size) {
  __shared__ float2 srcTmps[TPB * VPT / 2];
  __shared__ half2 dstTmps[TPB * VPT / 2];
  for (size_t tid = (threadIdx.x + blockIdx.x * blockDim.x) * VPT; tid < size;
       tid += blockDim.x * gridDim.x * VPT) {
    cudaReadBytes<VPT * sizeof(float)>(
        reinterpret_cast<void *>(srcTmps + threadIdx.x * VPT / 2),
        reinterpret_cast<void const *>(src + tid));
    __syncwarp();
    int sharedid = threadIdx.x / 32 * (32 * VPT / 2) + threadIdx.x % 32;
#pragma unroll
    for (int i = 0; i < VPT / 2; ++i) {
      dstTmps[sharedid + i * 32] =
          __float22half2_rn(srcTmps[sharedid + i * 32]);
    }
    __syncwarp();
    cudaReadBytes<VPT * sizeof(half)>(
        reinterpret_cast<void *>(dst + tid),
        reinterpret_cast<void const *>(dstTmps + threadIdx.x * VPT / 2));
  }
}

template <size_t TPB, size_t VPT>
__global__ void cudaCastHalf2Float(float *dst, half const *const src,
                                   const size_t size) {
  __shared__ half2 srcTmps[TPB * VPT / 2];
  __shared__ float2 dstTmps[TPB * VPT / 2];
  for (size_t tid = (threadIdx.x + blockIdx.x * blockDim.x) * VPT; tid < size;
       tid += blockDim.x * gridDim.x * VPT) {
    cudaReadBytes<VPT * sizeof(half)>(
        reinterpret_cast<void *>(srcTmps + threadIdx.x * VPT / 2),
        reinterpret_cast<void const *>(src + tid));
    __syncwarp();
    int sharedid = threadIdx.x / 32 * (32 * VPT / 2) + threadIdx.x % 32;
#pragma unroll
    for (int i = 0; i < VPT / 2; ++i) {
      dstTmps[sharedid + i * 32] = __half22float2(srcTmps[sharedid + i * 32]);
    }
    __syncwarp();
    cudaReadBytes<VPT * sizeof(float)>(
        reinterpret_cast<void *>(dst + tid),
        reinterpret_cast<void const *>(dstTmps + threadIdx.x * VPT / 2));
  }
}

template <typename T_OUT, typename T_IN>
MLIR_GPU_OPS_EXPORT void invokeCudaCast(T_OUT *dst, T_IN const *const src,
                                        const size_t size,
                                        hipStream_t stream) {
  cudaCast<T_OUT, T_IN, 256, 4><<<256, 256, 0, stream>>>(dst, src, size);
};

template <>
MLIR_GPU_OPS_EXPORT void
invokeCudaCast<half, float>(half *dst, float const *const src,
                            const size_t size, hipStream_t stream) {
  cudaCastFloat2Half<256, 4><<<256, 256, 0, stream>>>(dst, src, size);
}

template <>
MLIR_GPU_OPS_EXPORT void
invokeCudaCast<float, half>(float *dst, half const *const src,
                            const size_t size, hipStream_t stream) {
  cudaCastHalf2Float<256, 4><<<256, 256, 0, stream>>>(dst, src, size);
}